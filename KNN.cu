
#include <hip/hip_runtime.h>
#include<iostream>
#include<map>
#include<vector>
#include<stdio.h>
#include<cmath>
#include<cstdlib>
#include<algorithm>
#include<fstream>
 
using namespace std;
 
typedef string tLabel;
typedef float tData;
typedef pair<int,double>  PAIR;
const int MaxColLen = 10;
const int MaxRowLen = 10010;
const int test_data_num = 400;
ifstream fin;

float gpu_time = 0.0;
class KNN
{
private:
		tData dataSet[MaxRowLen][MaxColLen];
		tLabel labels[MaxRowLen];
		tData testData[MaxColLen];
		tData trainingData[3600][8];
		int rowLen;
		int colLen;
		int k;
		map<int,double> map_index_dis;
		map<tLabel,int> map_label_freq;
		double get_distance(tData *d1,tData *d2);
public:
		KNN(int k , int rowLen , int colLen , char *filename);
		void get_all_distance();
		tLabel get_max_freq_label();
		void auto_norm_data();
		void get_error_rate();
		void get_training_data();
		struct CmpByValue
		{
			bool operator() (const PAIR& lhs,const PAIR& rhs)
			{
				return lhs.second < rhs.second;
			}
		};
 
		~KNN();	
};
 
KNN::~KNN()
{
	fin.close();
	map_index_dis.clear();
	map_label_freq.clear();
}
 
KNN::KNN(int k , int row ,int col , char *filename)
{
	this->rowLen = row;
	this->colLen = col;
	this->k = k;
	
	fin.open(filename);
 
	if( !fin )
	{
		cout<<"can not open the file"<<endl;
		exit(0);
	}
 
	for(int i=0;i<rowLen;i++)
	{
		for(int j=0;j<colLen;j++)
		{
			fin>>dataSet[i][j];
		}
		fin>>labels[i];
	}
 
}
 
void KNN:: get_training_data()
{
	for(int i=test_data_num;i<rowLen;i++)
	{
		for(int j=0;j<colLen;j++)
		{
			trainingData[i-test_data_num][j] = dataSet[i][j];
		}
	}
}
 
void KNN:: get_error_rate()
{
	int i,j,count = 0;
	tLabel label;
 
	cout<<"the test data number is : "<<test_data_num<<endl;
 
	get_training_data();
 
	//get testing data and calculate
	for(i=0;i<test_data_num;i++)
	{
		for(j=0;j<colLen;j++)
		{
			testData[j] = dataSet[i][j];		
		}
		
		get_all_distance();
		label = get_max_freq_label();
		if( label!=labels[i] )
			count++;
		map_index_dis.clear();
		map_label_freq.clear();
	}
	cout<<"the error rate is = "<<(double)count/(double)test_data_num<<endl;
}
 
//global function
__global__ void cal_dis(tData *train_data,tData *test_data,tData* dis,int pitch,int N , int D)
{
	int tid = blockIdx.x;
	if(tid<N)
	{
		tData temp = 0;
		tData sum = 0;
		for(int i=0;i<D;i++)
		{
			temp = *( (tData*)( (char*)train_data+tid*pitch  )+i ) - test_data[i];
			sum += temp * temp;
		}
		dis[tid] = sum;
	}
}
 
//Parallel calculate the distance
void KNN:: get_all_distance()
{
	// cal GPU run time
	// cudaEvent_t gpu_start, gpu_stop;
	// cudaEventCreate(&gpu_start);
	// cudaEventCreate(&gpu_stop);
	// cudaEventRecord(gpu_start, 0);

	int height = rowLen - test_data_num;
	tData *distance = new tData[height];
	tData *d_train_data,*d_test_data,*d_dis;
	size_t pitch_d ;
	size_t pitch_h = colLen * sizeof(tData);
	//allocate memory on GPU
	hipMallocPitch( &d_train_data,&pitch_d,colLen*sizeof(tData),height);
	hipMalloc( &d_test_data,colLen*sizeof(tData) );
	hipMalloc( &d_dis, height*sizeof(tData) );
 
	hipMemset( d_train_data,0,height*colLen*sizeof(tData) );
	hipMemset( d_test_data,0,colLen*sizeof(tData) );
	hipMemset( d_dis , 0 , height*sizeof(tData) );
 
	//copy training and testing data from host to device
	hipMemcpy2D( d_train_data,pitch_d,trainingData,pitch_h,colLen*sizeof(tData),height,hipMemcpyHostToDevice);
	hipMemcpy( d_test_data,testData,colLen*sizeof(tData),hipMemcpyHostToDevice);
	//calculate the distance
	cal_dis<<<height,1>>>( d_train_data,d_test_data,d_dis,pitch_d,height,colLen );
	//copy distance data from device to host
	hipMemcpy( distance,d_dis,height*sizeof(tData),hipMemcpyDeviceToHost);
	
	// cudaEventRecord(gpu_stop, 0);
	// cudaEventSynchronize(gpu_stop);

	// float timestamp;
	// cudaEventElapsedTime(&timestamp, gpu_start, gpu_stop);
	// gpu_time += timestamp;

	// cudaEventDestroy(gpu_start);
	// cudaEventDestroy(gpu_stop);
	int i;
	for( i=0;i<rowLen-test_data_num;i++ )
	{
		map_index_dis[i+test_data_num] = distance[i];
	}
 
}
 
tLabel KNN:: get_max_freq_label()
{
	vector<PAIR> vec_index_dis( map_index_dis.begin(),map_index_dis.end() );
	sort(vec_index_dis.begin(),vec_index_dis.end(),CmpByValue());
 
	for(int i=0;i<k;i++)
	{
		/*
		cout<<"the index = "<<vec_index_dis[i].first<<" the distance = "<<vec_index_dis[i].second<<" the label = "<<labels[ vec_index_dis[i].first ]<<" the coordinate ( ";
		int j;
		for(j=0;j<colLen-1;j++)
		{
			cout<<dataSet[ vec_index_dis[i].first ][j]<<",";
		}
		cout<<dataSet[ vec_index_dis[i].first ][j]<<" )"<<endl;
		*/
		map_label_freq[ labels[ vec_index_dis[i].first ]  ]++;
	}
 
	map<tLabel,int>::const_iterator map_it = map_label_freq.begin();
	tLabel label;
	int max_freq = 0;
	while( map_it != map_label_freq.end() )
	{
		if( map_it->second > max_freq )
		{
			max_freq = map_it->second;
			label = map_it->first;
		}
		map_it++;
	}
	// cout<<"The test data belongs to the "<<label<<" label"<<endl;
	return label;
}
 
void KNN::auto_norm_data()
{
	tData maxa[colLen] ;
	tData mina[colLen] ;
	tData range[colLen] ;
	int i,j;
 
	for(i=0;i<colLen;i++)
	{
		maxa[i] = max(dataSet[0][i],dataSet[1][i]);
		mina[i] = min(dataSet[0][i],dataSet[1][i]);
	}
 
	for(i=2;i<rowLen;i++)
	{
		for(j=0;j<colLen;j++)
		{
			if( dataSet[i][j]>maxa[j] )
			{
				maxa[j] = dataSet[i][j];
			}
			else if( dataSet[i][j]<mina[j] )
			{
				mina[j] = dataSet[i][j];
			}
		}
	}
 
	for(i=0;i<colLen;i++)
	{
		range[i] = maxa[i] - mina[i] ; 
		//normalize the test data set
		testData[i] = ( testData[i] - mina[i] )/range[i] ;
	}
 
	//normalize the training data set
	for(i=0;i<rowLen;i++)
	{
		for(j=0;j<colLen;j++)
		{
			dataSet[i][j] = ( dataSet[i][j] - mina[j] )/range[j];
		}
	}
}
 
int main(int argc , char** argv)
{
	int k,row,col;
	char *filename;
	
	if( argc!=5 )
	{
		cout<<"The input should be like this : ./a.out k row col filename"<<endl;
		exit(1);
	}
 
	k = atoi(argv[1]);
	row = atoi(argv[2]);
	col = atoi(argv[3]);
	filename = argv[4];
 
	KNN knn(k,row,col,filename);

	knn.auto_norm_data();
	knn.get_error_rate();

	printf("GPU:Time used:%.2f ms\n", gpu_time);
	return 0;
}